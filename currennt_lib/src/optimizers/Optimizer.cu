#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#include "Optimizer.hpp"
#include "../layers/TrainableLayer.hpp"
#include "../layers/BinaryClassificationLayer.hpp"
#include "../layers/MulticlassClassificationLayer.hpp"
#include "../Configuration.hpp"
#include "../helpers/JsonClasses.hpp"
#include "../helpers/getRawPointer.cuh"

#include <limits>

#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

namespace thrust {
    template <typename floatT>
    struct elasticnet_dfunctor {
        const floatT a;
        const floatT b;
        elasticnet_dfunctor(floatT _a, floatT _b) : a(_a), b(_b) {}

        __host__ __device__
            floatT operator()(const floatT& x, const floatT& y) const {
                return a * x + copysign(b, x) + y;
            }
    };

    template <typename floatT>
    struct elasticnet_functor {
        const floatT a;
        const floatT b;
        elasticnet_functor(floatT _a, floatT _b) : a(_a), b(_b) {}

        __host__ __device__
            floatT operator()(const floatT& x) const {
                return a * x * x + b * fabs(x);
            }
    };

}

namespace optimizers {
    template <typename TDevice>
    real_t Optimizer<TDevice>::_processDataSet(data_sets::DataSet &ds, bool calcWeightUpdates, real_t *classError)
    {
        // process all data set fractions
        real_t error = 0;
        *classError = (real_t) ds.totalTimesteps();

        boost::shared_ptr<data_sets::DataSetFraction> frac;
        bool firstFraction = true;
        while ((frac = ds.getNextFraction())) {
            // compute forward pass and calculate the error
            m_neuralNetwork.loadSequences(*frac);
            m_neuralNetwork.computeForwardPass();
            error += m_neuralNetwork.calculateError();

            if (dynamic_cast<layers::BinaryClassificationLayer<TDevice>*>(&m_neuralNetwork.postOutputLayer()))
                *classError -= (real_t)static_cast<layers::BinaryClassificationLayer<TDevice>&>(m_neuralNetwork.postOutputLayer()).countCorrectClassifications();
            if (dynamic_cast<layers::MulticlassClassificationLayer<TDevice>*>(&m_neuralNetwork.postOutputLayer()))
                *classError -= (real_t)static_cast<layers::MulticlassClassificationLayer<TDevice>&>(m_neuralNetwork.postOutputLayer()).countCorrectClassifications();
            
            if (calcWeightUpdates) {
                // weight noise:
                std::vector<Cpu::real_vector> origWeights(m_neuralNetwork.layers().size());
                if (Configuration::instance().weightNoiseSigma() > 0.0) {
                    for (size_t i = 1; i < m_neuralNetwork.layers().size()-1; ++i) {
                        layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(m_neuralNetwork.layers()[i].get());
                        if (layer) {
                            origWeights[i] = layer->weights();
                            layer->injectWeightNoise(Configuration::instance().weightNoiseSigma());
                        }
                    }
                }
                // compute the backward pass and accumulate the weight updates
                m_neuralNetwork.computeBackwardPass();

                for (size_t i = 1; i < m_neuralNetwork.layers().size()-1; ++i) {
                    layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(m_neuralNetwork.layers()[i].get());
                    if (!layer)
                        continue;

                    if (!firstFraction && !Configuration::instance().hybridOnlineBatch())
                        thrust::transform(layer->weightUpdates().begin(), layer->weightUpdates().end(), m_curWeightUpdates[i].begin(), m_curWeightUpdates[i].begin(), thrust::plus<real_t>());
                    else
                    	thrust::copy(layer->weightUpdates().begin(), layer->weightUpdates().end(), m_curWeightUpdates[i].begin());

                    // restore old weights before update in case of weight noise
                    if (Configuration::instance().weightNoiseSigma() > 0.0)
                        thrust::copy(origWeights[i].begin(), origWeights[i].end(), layer->weights().begin());
                }

                // update weights for hybrid online/batch learning
                if (Configuration::instance().hybridOnlineBatch()){
                    _updateWeights(error);
                }
            }

            firstFraction = false;
        }

        // Add elastic-net penalty terms
        for (size_t i = 1; i < m_neuralNetwork.layers().size()-1; ++i) {
            layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(m_neuralNetwork.layers()[i].get());
            if (!layer)
                continue;
            thrust::transform(layer->weights().begin(), layer->weights().end(), m_curWeightUpdates[i].begin(), m_curWeightUpdates[i].begin(), thrust::elasticnet_dfunctor<real_t>(m_alpha, m_beta));
            error += thrust::transform_reduce(layer->weights().begin(), layer->weights().end(), thrust::elasticnet_functor<real_t>(m_alpha, m_beta), 0.0, thrust::plus<real_t>());
        }

        // normalize the errors
        error /= ds.totalSequences();
        *classError /= (real_t)ds.totalTimesteps();

        // update weights for batch learning
        if (calcWeightUpdates && !Configuration::instance().hybridOnlineBatch()){
            _updateWeights(error);
        }

        return error;
    }

    template <typename TDevice>
    void Optimizer<TDevice>::_exportWeights(const helpers::JsonDocument &jsonDoc, const char *arrayName, const std::vector<real_vector> &weights)
    {
        rapidjson::Value weightsArray(rapidjson::kArrayType);
        weightsArray.Reserve((rapidjson::SizeType)weights.size(), jsonDoc->GetAllocator());

        for (size_t i = 0; i < weights.size(); ++i) {
            rapidjson::Value v(rapidjson::kArrayType);
            Cpu::real_vector w = weights[i];
            v.Reserve((rapidjson::SizeType)w.size(), jsonDoc->GetAllocator());
            for (size_t j = 0; j < w.size(); ++j)
                v.PushBack(w[j], jsonDoc->GetAllocator());
            weightsArray.PushBack(v, jsonDoc->GetAllocator());
        }

        jsonDoc->AddMember(arrayName, weightsArray, jsonDoc->GetAllocator());
    }

    template <typename TDevice>
    void Optimizer<TDevice>::_importWeights(const helpers::JsonDocument &jsonDoc, const char *arrayName, std::vector<real_vector> *weights)
    {
        if (!jsonDoc->HasMember(arrayName) || !(*jsonDoc)[arrayName].IsArray())
            throw std::runtime_error(std::string("Array '") + arrayName + "' is missing or has the wrong type");

        if ((*jsonDoc)[arrayName].Size() != (rapidjson::SizeType)weights->size())
            throw std::runtime_error(std::string("Array '") + arrayName + "' has a wrong size");

        int i = 0;
        for (rapidjson::Value::ConstValueIterator it = (*jsonDoc)[arrayName].Begin(); it != (*jsonDoc)[arrayName].End(); ++it) {
            if (!it->IsArray())
                throw std::runtime_error(std::string("Object in '") + arrayName + "' is not an array");
            if (it->Size() != (rapidjson::SizeType)(*weights)[i].size())
                throw std::runtime_error(std::string("Subarray in '") + arrayName + "' has a wrong size");

            Cpu::real_vector w;
            w.reserve(it->Size());
            for (rapidjson::Value::ConstValueIterator it2 = it->Begin(); it2 != it->End(); ++it2)
                w.push_back((real_t)it2->GetDouble());

            (*weights)[i] = w;

            ++i;
        }
    }

    template <typename TDevice>
    void Optimizer<TDevice>::storeWeights(std::vector<real_vector>& x)
    {
        for (size_t i = 1; i < m_neuralNetwork.layers().size() - 1; ++i) {
            layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(m_neuralNetwork.layers()[i].get());
            if (layer){
            	thrust::copy(layer->weights().begin(), layer->weights().end(), x[i].begin());
            }
        }
    }

    template <typename TDevice>
    void Optimizer<TDevice>::_restoreWeights(const std::vector<real_vector>& x)
    {
        for (size_t i = 1; i < m_neuralNetwork.layers().size() - 1; ++i) {
            layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(m_neuralNetwork.layers()[i].get());
            if (layer){
            	thrust::copy(x[i].begin(), x[i].end(), layer->weights().begin());
            }
        }
    }

    template <typename TDevice>
    void Optimizer<TDevice>::resetWeights(void)
    {
        _restoreWeights(m_lastWeights);
    }


    template <typename TDevice>
    NeuralNetwork<TDevice>& Optimizer<TDevice>::_neuralNetwork()
    {
        return m_neuralNetwork;
    }

    template <typename TDevice>
    const std::vector<typename Optimizer<TDevice>::real_vector>& Optimizer<TDevice>::_curWeightUpdates() const
    {
        return m_curWeightUpdates;
    }

    template <typename TDevice>
    Optimizer<TDevice>::Optimizer(NeuralNetwork<TDevice> &neuralNetwork, data_sets::DataSet &trainingSet, 
                                   data_sets::DataSet &validationSet, data_sets::DataSet &testSet,
                                   int maxEpochs, int maxEpochsNoBest, int validateEvery, int testEvery,
                                   real_t alpha, real_t beta)
        : m_neuralNetwork             (neuralNetwork)
        , m_trainingSet               (trainingSet)
        , m_validationSet             (validationSet)
        , m_testSet                   (testSet)
        , m_maxEpochs                 (maxEpochs)
        , m_maxEpochsNoBest           (maxEpochsNoBest)
        , m_validateEvery             (validateEvery)
        , m_testEvery                 (testEvery)
        , m_finished                  (false)
        , m_curEpoch                  (0)
        , m_epochsSinceLowestError    (0)
        , m_lowestValidationError     (std::numeric_limits<real_t>::max())
        , m_curTrainingError          (std::numeric_limits<real_t>::max())
        , m_curValidationError        (std::numeric_limits<real_t>::max())
        , m_curTestError              (std::numeric_limits<real_t>::max())
        , m_curValidationClassError   (0)
        , m_curTrainingClassError     (0)
        , m_curTestClassError         (0)
        , m_alpha                     (alpha)
        , m_beta                      (beta)
    {
        // initialize the last and best weights vectors
        m_lastWeights.resize(m_neuralNetwork.layers().size());
        m_bestWeights.resize(m_neuralNetwork.layers().size());
        for (size_t i = 1; i < m_neuralNetwork.layers().size()-1; ++i) {
       	    layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(m_neuralNetwork.layers()[i].get());
            if (layer){
                m_lastWeights[i] = layer->weights();
                m_bestWeights[i] = layer->weights();
            }
        }

        // initialize the current weight updates vectors
        m_curWeightUpdates = m_bestWeights;
    }

    template <typename TDevice>
    Optimizer<TDevice>::~Optimizer()
    {
    }

    template <typename TDevice>
    bool Optimizer<TDevice>::finished() const
    {
        return m_finished;
    }

    template <typename TDevice>
    int Optimizer<TDevice>::currentEpoch() const
    {
        return m_curEpoch;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::lowestValidationError() const
    {
        return m_lowestValidationError;
    }

    template <typename TDevice>
    int Optimizer<TDevice>::epochsSinceLowestValidationError() const
    {
        return m_epochsSinceLowestError;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::curTrainingError() const
    {
        return m_curTrainingError;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::curValidationError() const
    {
        return m_curValidationError;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::curTestError() const
    {
        return m_curTestError;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::curTrainingClassError() const
    {
        return m_curTrainingClassError;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::curValidationClassError() const
    {
        return m_curValidationClassError;
    }

    template <typename TDevice>
    real_t Optimizer<TDevice>::curTestClassError() const
    {
        return m_curTestClassError;
    }

    template <typename TDevice>
    bool Optimizer<TDevice>::train()
    {
        if (!m_finished) {
            ++m_curEpoch;

            // train one epoch and update the weights
            m_curTrainingError = _processDataSet(m_trainingSet, true, &m_curTrainingClassError);

            // calculate the validation error and store the weights if we a new lowest error
            if (!m_validationSet.empty() && m_curEpoch % m_validateEvery == 0) {
                m_curValidationError = _processDataSet(m_validationSet, false, &m_curValidationClassError);
                
                if (m_curValidationError < m_lowestValidationError) {
                    m_lowestValidationError  = m_curValidationError;
                    m_epochsSinceLowestError = 0;

                    storeWeights(m_bestWeights);
                }
                else {
                    m_epochsSinceLowestError += m_validateEvery;
                }
            }
            else if (m_validationSet.empty()) {
                m_epochsSinceLowestError = 0;
                storeWeights(m_bestWeights);
            }

            // calculate the test error
            if (!m_testSet.empty() && m_curEpoch % m_testEvery == 0)
                m_curTestError = _processDataSet(m_testSet, false, &m_curTestClassError);

            // check if we did not get a new lowest error for some training epochs 
            // or if we reached the maximum number of training epochs
            if (m_epochsSinceLowestError >= m_maxEpochsNoBest || (m_maxEpochs >= 0 && m_curEpoch >= m_maxEpochs)) {
                _restoreWeights(m_bestWeights);
                m_finished = true;
            }
        }

        return m_finished;
    }

    template <typename TDevice>
    void Optimizer<TDevice>::exportState(const helpers::JsonDocument &jsonDoc) const
    {
        jsonDoc->AddMember("optimizer_finished",                   m_finished,                jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_epoch",                  m_curEpoch,                jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_epochs_since_lowest_error",  m_epochsSinceLowestError,  jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_lowest_validation_error",    m_lowestValidationError,   jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_training_error",         m_curTrainingError,        jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_validation_error",       m_curValidationError,      jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_test_error",             m_curTestError,            jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_training_class_error",   m_curTrainingClassError,   jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_validation_class_error", m_curValidationClassError, jsonDoc->GetAllocator());
        jsonDoc->AddMember("optimizer_cur_test_class_error",       m_curTestClassError,       jsonDoc->GetAllocator());

        _exportWeights(jsonDoc, "optimizer_best_weights", m_bestWeights);
    }

    template <typename TDevice>
    void Optimizer<TDevice>::importState(const helpers::JsonDocument &jsonDoc)
    {
        m_finished                = helpers::checkedJsonGet<bool  >(*jsonDoc, "optimizer_finished");
        m_curEpoch                = helpers::checkedJsonGet<int   >(*jsonDoc, "optimizer_cur_epoch");
        m_epochsSinceLowestError  = helpers::checkedJsonGet<int   >(*jsonDoc, "optimizer_epochs_since_lowest_error");
        m_lowestValidationError   = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_lowest_validation_error");
        m_curTrainingError        = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_cur_training_error");
        m_curValidationError      = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_cur_validation_error");
        m_curTestError            = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_cur_test_error");
        m_curTrainingClassError   = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_cur_training_class_error");
        m_curValidationClassError = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_cur_validation_class_error");
        m_curTestClassError       = helpers::checkedJsonGet<real_t>(*jsonDoc, "optimizer_cur_test_class_error");

        _importWeights(jsonDoc, "optimizer_best_weights", &m_bestWeights);
    }


    // explicit template instantiations
    template class Optimizer<Cpu>;
    template class Optimizer<Gpu>;

} // namespace optimizers
